
#include <hip/hip_runtime.h>
/*
Copyright (c) 2014 Matthew Simons

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#define colIndex(i,j,ld) (((j)*(ld))+(i))

/**
 *  implementation of the Hillis & Steel inclusive scan reduction method
 */
__device__ void reduce(double *v, int length)
{
	extern __shared__ double sdata[];

	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	int index2 = index * 2 + 1;

	if(index2 < length)
	{
		sdata[index] = v[index2] + v[index2 - 1];
	}
	__syncthreads();

	v[index] = sdata[index];
}

extern "C"
__global__ void scale(double *v1, int length, double factor)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v1[index] = v1[index] * factor;
	}
}

extern "C"
__global__ void power(double *v1, int length, double factor)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v1[index] = pow(v1[index], factor);
	}
}

extern "C"
__global__ void innerProduct(double *v1, double *v2, int length, double *ans)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	extern __shared__ double products[];

	// Find the product of each value
	if(index < length)
	{
		products[index] = v1[index] * v2[index];
		//printf("product of %f * %f = %f\n", v1[index], v2[index], products[index]);
	}
	__syncthreads();

	// Sum all the products
	while(length != 0)
	{
		reduce(products, length);
		length = length / 2;
		__syncthreads();
	}

	// First thread copies answer to global mem
	if(index == 0) {
		ans[0] = products[0];
	}
}

extern "C"
__global__ void multiply(double *m1, double *m2, int width, int height, double *result)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	//int corresponding =

	//multiply each row and column and reduce atomic add maybe

}



extern "C"
__global__ void vectorDot(double *v1, double *v2, int length, double *v3)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v3[index] = v1[index] * v2[index];
	}
	// returns *v3
}
