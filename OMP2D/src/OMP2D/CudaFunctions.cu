#include "hip/hip_runtime.h"
extern "C"
__global__ void scale(double *v1, int length, double factor)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v1[index] = v1[index] * factor;
	}
}

extern "C"
__global__ void innerProduct(double *v1, double *v2, int length)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v1[index] = v1[index] * v2[index];
	}
	__syncthreads();

	reduce(v1, length);
}



extern "C"
__global__ void vectorDot(double *v1, double *v2, int length, double *v3)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v3[index] = v1[index] * v2[index];
	}
	// returns *v3
}

__device__ void reduce(double *v1, int length)
{
	__shared__ double sdata[1024];
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	int index2 = index * 2 + 1;
	//int index2 = length/2+index;
	if(index2 < length)
	{
		sdata[index] = v1[index2] + v1[index2 - 1];
		//v1[index] = v1[index] + v1[index2];
	}
	__syncthreads();
	v1[index] = sdata[index];
	//printf("%f ", v1[index]);
}

extern "C"
__global__ void shmem_reduce_kernel(double *d_out, double *d_in)
{

    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ double sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];

    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];

        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

extern "C"
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // do reduction in global mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            d_in[myId] += d_in[myId + s];
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = d_in[myId];
    }
}
