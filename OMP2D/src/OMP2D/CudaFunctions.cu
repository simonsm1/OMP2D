
#include <hip/hip_runtime.h>
#define colIndex(i,j,ld) (((j)*(ld))+(i))

/**
 *  implementation of the Hillis & Steel inclusive scan reduction method
 */
__device__ void reduce(double *v, int length)
{
	extern __shared__ double sdata[];

	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	int index2 = index * 2 + 1;

	if(index2 < length)
	{
		sdata[index] = v[index2] + v[index2 - 1];
	}
	__syncthreads();

	v[index] = sdata[index];
}

extern "C"
__global__ void scale(double *v1, int length, double factor)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v1[index] = v1[index] * factor;
	}
}

extern "C"
__global__ void power(double *v1, int length, double factor)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v1[index] = pow(v1[index], factor);
	}
}

extern "C"
__global__ void innerProduct(double *v1, double *v2, int length, double *ans)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	extern __shared__ double products[];

	// Find the product of each value
	if(index < length)
	{
		products[index] = v1[index] * v2[index];
		//printf("product of %f * %f = %f\n", v1[index], v2[index], products[index]);
	}
	__syncthreads();

	// Sum all the products
	while(length != 0)
	{
		reduce(products, length);
		length = length / 2;
		__syncthreads();
	}

	// First thread copies answer to global mem
	if(index == 0) {
		ans[0] = products[0];
	}
}

extern "C"
__global__ void multiply(double *m1, double *m2, int width, int height, double *result)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	//int corresponding =

	//multiply each row and column and reduce atomic add maybe

}



extern "C"
__global__ void vectorDot(double *v1, double *v2, int length, double *v3)
{
	int index = blockIdx.x*threadIdx.x + threadIdx.x;
	if(index < length)
	{
		v3[index] = v1[index] * v2[index];
	}
	// returns *v3
}
